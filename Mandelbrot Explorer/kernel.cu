#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <stdio.h>
#include <cstdint>
#include <mpir.h>
#include "string.h"

// TODO: Repair mode, select an area to re-render at a higher sample rate

#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %d %s:%d: '%s'\n",e,__FILE__,__LINE__,hipGetErrorString(e));           \
   /*exit(0);*/ \
 }                                                                 \
}
typedef struct mpfloat {
	uint8_t *digits;
	uint64_t decimal;
	bool negative;
} mpfloat;

__device__ uint32_t mpfloat_precision;


// Device Functions
__device__ void mpfloat_init(mpfloat *multifloat);
__device__ void mpfloat_free(mpfloat multifloat);
__device__ void mpfloat_clear(mpfloat *multifloat);

__device__ void mpfloat_set_uint32(uint32_t a, mpfloat *result);
__device__ void mpfloat_set_mpfloat(mpfloat *a, mpfloat b);

__device__ int8_t mpfloat_cmp(mpfloat a, mpfloat b);
__device__ int8_t mpfloat_cmp_ui(mpfloat a, uint32_t b);

__device__ void mpfloat_pack_left(mpfloat *multifloat);
__device__ void mpfloat_shift_right_insert(mpfloat *multifloat, uint8_t digit);

__device__ void mpfloat_get_extent(mpfloat a, uint64_t *extent);
__device__ uint32_t mpfloat_get_num_digits(mpfloat multifloat);
__device__ uint32_t mpfloat_get_num_decimals(mpfloat multifloat);

__device__ void mpfloat_add(mpfloat a, mpfloat b, mpfloat *result);
__device__ void mpfloat_sub(mpfloat a, mpfloat b, mpfloat *result);

__device__ void mpfloat_mul(mpfloat a, mpfloat b, mpfloat *result);
__device__ void mpfloat_mul_ui(mpfloat a, uint32_t b, mpfloat *result);

__device__ void mpfloat_to_str(mpfloat multifloat, char *buffer, uint32_t length);
__device__ void mpfloat_to_double(mpfloat multifloat, double *result);

// Host Functions
void mpfloat_sub_host(mpfloat a, mpfloat b, mpfloat *result);
void mpfloat_add_host(mpfloat a, mpfloat b, mpfloat *result);
void mpfloat_mul_host(mpfloat a, mpfloat b, mpfloat *result);
void mpfloat_mul_ui_host(mpfloat a, uint32_t b, mpfloat *result);
int8_t mpfloat_cmp_host(mpfloat a, mpfloat b);

void mpfloat_set_mpfloat_host(mpfloat *a, mpfloat b);
void mpfloat_set_mpir(mpfloat *multifloat, mpf_t mpirfloat);
void mpfloat_set_str_host(mpfloat *multifloat, char *floatString);
void mpfloat_set_double_host(double a, mpfloat *result);

void mpfloat_set_precision(uint32_t precision);


void mpfloat_get_extent_host(mpfloat a, uint64_t *extent);
uint32_t mpfloat_get_num_decimals_host(mpfloat multifloat);
uint32_t mpfloat_get_num_digits_host(mpfloat multifloat);

void mpfloat_init_host(mpfloat *multifloat);
void mpfloat_free_host(mpfloat multifloat);
void mpfloat_clear_host(mpfloat *multifloat);
void mpfloat_to_str_host(mpfloat multifloat, char *buffer, uint32_t length);

void mpfloat_test_add_host();

struct test_data
{
	char *A;
	char *B;
	char *Solution;
};

#define NUM_ADD_TEST_INPUTS 10
struct test_data add_test_input[NUM_ADD_TEST_INPUTS] = {
	{ "9.99999999","0.00000001","10" },
	{ "0","1.234","1.234" },
	{ "9.8634","0.9463","10.8097" },
	{ "99.8634","0.9463","100.8097" },
	{ "0","0.9463","0.9463" },
	{ "1","2","3" },
	{ "1","0.0000002","1.0000002" },
	{ "1","0","1" },
	{ "0.99999999","0.00000001","1" },
	{ "0","0","0" }
};

void mpfloat_test_add_host()
{
	char ABuff[100];
	char BBuff[100];
	char ResultBuff[100];
	char SolutionBuff[100];
	mpfloat A;
	mpfloat_init_host(&A);
	mpfloat B;
	mpfloat_init_host(&B);
	mpfloat Result;
	mpfloat_init_host(&Result);
	mpfloat Solution;
	mpfloat_init_host(&Solution);
	bool Correct;
	bool GotFailure = false;
	int testsPassed = 0;

	fprintf(stdout, "\nADD TEST\n----------------\n");

	for (uint32_t TestInputIter = 0; TestInputIter < NUM_ADD_TEST_INPUTS; TestInputIter++)
	{
		GotFailure = false;
		mpfloat_clear_host(&Result);
		mpfloat_set_str_host(&A, add_test_input[TestInputIter].A);
		mpfloat_set_str_host(&B, add_test_input[TestInputIter].B);
		mpfloat_set_str_host(&Solution, add_test_input[TestInputIter].Solution);
		mpfloat_add_host(A, B, &Result);
		Correct = (mpfloat_cmp_host(Result, Solution) == 0);
		if (!Correct)
		{
			GotFailure = true;
		}
		mpfloat_to_str_host(A, ABuff, 98);
		mpfloat_to_str_host(B, BBuff, 98);
		mpfloat_to_str_host(Result, ResultBuff, 98);
		mpfloat_to_str_host(Solution, SolutionBuff, 98);
		fprintf(stdout, "%s + %s\nExpected: %s\nActual:   %s \n%s\n", ABuff, BBuff, SolutionBuff, ResultBuff, Correct ? "PASS":"FAIL");
		mpfloat_clear_host(&Result);
		mpfloat_add_host(B, A, &Result);
		Correct = (mpfloat_cmp_host(Result, Solution) == 0);
		if (!Correct)
		{
			GotFailure = true;
		}
		mpfloat_to_str_host(Result, ResultBuff, 98);
		fprintf(stdout, "%s + %s\nExpected: %s\nActual:   %s \n%s\n", BBuff, ABuff, SolutionBuff, ResultBuff, Correct ? "PASS" : "FAIL");
		fprintf(stdout, "Test Case %u : %s\n", TestInputIter, GotFailure ? "FAIL" : "PASS");
		fprintf(stdout, "---------------------------\n\n");
		if (!GotFailure)
		{
			testsPassed++;
		}
	}
	fprintf(stdout, "Passed %d of %d Tests. Score: %f\n\n", testsPassed, NUM_ADD_TEST_INPUTS,100*((double)testsPassed/(double)NUM_ADD_TEST_INPUTS));

}

#define NUM_SUB_TEST_INPUTS 10
struct test_data sub_test_input[NUM_ADD_TEST_INPUTS] = {
	{"10","9.99999999","0.00000001"},
	{ "0","1.234","-1.234" },
	{ "9.8634","0.9463","10.8097" },
	{ "99.8634","0.9463","100.8097" },
	{ "0","0.9463","0.9463" },
	{ "1","2","3" },
	{ "1","0.0000002","1.0000002" },
	{ "1","0","1" },
	{ "0.99999999","0.00000001","1" },
	{ "0","0","0" }
};

void mpfloat_test_sub_host()
{
	char ABuff[100];
	char BBuff[100];
	char ResultBuff[100];
	char SolutionBuff[100];
	mpfloat A;
	mpfloat_init_host(&A);
	mpfloat B;
	mpfloat_init_host(&B);
	mpfloat Result;
	mpfloat_init_host(&Result);
	mpfloat Solution;
	mpfloat_init_host(&Solution);
	bool Correct;
	bool GotFailure = false;
	int testsPassed = 0;

	fprintf(stdout, "\nSUB TEST\n----------------\n");

	for (uint32_t TestInputIter = 0; TestInputIter < NUM_SUB_TEST_INPUTS; TestInputIter++)
	{
		GotFailure = false;
		mpfloat_clear_host(&Result);
		mpfloat_set_str_host(&A, sub_test_input[TestInputIter].A);
		mpfloat_set_str_host(&B, sub_test_input[TestInputIter].B);
		mpfloat_set_str_host(&Solution, sub_test_input[TestInputIter].Solution);
		mpfloat_sub_host(A, B, &Result);
		Correct = (mpfloat_cmp_host(Result, Solution) == 0);
		if (!Correct)
		{
			GotFailure = true;
		}
		mpfloat_to_str_host(A, ABuff, 98);
		mpfloat_to_str_host(B, BBuff, 98);
		mpfloat_to_str_host(Result, ResultBuff, 98);
		mpfloat_to_str_host(Solution, SolutionBuff, 98);
		fprintf(stdout, "%s - %s\nExpected: %s\nActual:   %s \n%s\n", ABuff, BBuff, SolutionBuff, ResultBuff, Correct ? "PASS" : "FAIL");
		fprintf(stdout, "Test Case %u : %s\n", TestInputIter, GotFailure ? "FAIL" : "PASS");
		fprintf(stdout, "---------------------------\n\n");
		if (!GotFailure)
		{
			testsPassed++;
		}
	}
	fprintf(stdout, "Passed %d of %d Tests. Score: %f\n\n", testsPassed, NUM_SUB_TEST_INPUTS, 100 * ((double)testsPassed / (double)NUM_SUB_TEST_INPUTS));

}

#define NUM_MUL_TEST_INPUTS 9
struct test_data mul_test_input[NUM_MUL_TEST_INPUTS] = {
	{ "-10","1","-10" },
	{ "-1","1","-1" },
	{ "-10.001","-0.1","1.0001" },
	{ "2","3","6" },
	{ "-0.022","-3.998","0.087956" },
	{ "6.983","7.423","51.834809" },
	{ "0.55487","28.513873","15.82149271151" },
	{"1","1","1"},
	{"1","0","0"}
};

void mpfloat_test_mul_host()
{
	char ABuff[100];
	char BBuff[100];
	char ResultBuff[100];
	char SolutionBuff[100];
	mpfloat A;
	mpfloat_init_host(&A);
	mpfloat B;
	mpfloat_init_host(&B);
	mpfloat Result;
	mpfloat_init_host(&Result);
	mpfloat Solution;
	mpfloat_init_host(&Solution);
	bool Correct;
	bool GotFailure = false;
	int testsPassed = 0;

	fprintf(stdout, "\nMUL TEST\n----------------\n");

	for (uint32_t TestInputIter = 0; TestInputIter < NUM_MUL_TEST_INPUTS; TestInputIter++)
	{
		GotFailure = false;
		mpfloat_clear_host(&Result);
		mpfloat_set_str_host(&A, mul_test_input[TestInputIter].A);
		mpfloat_set_str_host(&B, mul_test_input[TestInputIter].B);
		mpfloat_set_str_host(&Solution, mul_test_input[TestInputIter].Solution);
		mpfloat_mul_host(A, B, &Result);
		Correct = (mpfloat_cmp_host(Result, Solution) == 0);
		if (!Correct)
		{
			GotFailure = true;
		}
		mpfloat_to_str_host(A, ABuff, 98);
		mpfloat_to_str_host(B, BBuff, 98);
		mpfloat_to_str_host(Result, ResultBuff, 98);
		mpfloat_to_str_host(Solution, SolutionBuff, 98);
		fprintf(stdout, "%s * %s\nExpected: %s\nActual:   %s \n%s\n", ABuff, BBuff, SolutionBuff, ResultBuff, Correct ? "PASS" : "FAIL");
		fprintf(stdout, "Test Case %u : %s\n", TestInputIter, GotFailure ? "FAIL" : "PASS");
		fprintf(stdout, "---------------------------\n\n");
		if (!GotFailure)
		{
			testsPassed++;
		}
	}
	fprintf(stdout, "Passed %d of %d Tests. Score: %f\n\n", testsPassed, NUM_MUL_TEST_INPUTS, 100 * ((double)testsPassed / (double)NUM_MUL_TEST_INPUTS));

}

void mpfloat_set_precision(uint32_t precision)
{
	mpfloat_precision = precision;
	hipMemcpyToSymbol(HIP_SYMBOL("mpfloat_precision"), &precision, 0, 0, hipMemcpyHostToDevice);
}

__device__ void mpfloat_free(mpfloat multifloat)
{
	free(multifloat.digits);
}

void mpfloat_free_host(mpfloat multifloat)
{
	hipFree(multifloat.digits);
}

void mpfloat_clear_host(mpfloat *multifloat)
{
	for (uint32_t digitIter = 0; digitIter < mpfloat_precision; digitIter++)
	{
		multifloat->digits[digitIter] = 0;
	}
	multifloat->decimal = 0;
	multifloat->negative = false;
}

void mpfloat_init_host(mpfloat *multifloat)
{
	//multifloat->digits = new uint8_t[mpfloat_precision]();
	hipError_t status = hipMallocManaged(&multifloat->digits, mpfloat_precision * sizeof(uint8_t));
	if (status == 0)
	{
		mpfloat_clear_host(multifloat);
	}
	else
	{
		fprintf(stdout, "fuck\n");
	}
}

void mpfloat_set_str_host(mpfloat *multifloat, char *floatString)
{
	uint32_t mpfloatDigitIter = 0;
	uint32_t charDigitIter;
	uint32_t stringLength = strlen(floatString);
	bool FoundDecimal = false;
	bool NonZeroLeadingDigit = false;

	mpfloat_clear_host(multifloat);
	if (stringLength == 1 && floatString[0] == '0')
	{
		return;
	}

	for (charDigitIter = 0; charDigitIter < stringLength; charDigitIter++)
	{
		if (charDigitIter == 0 && floatString[charDigitIter] == '-')
		{
			multifloat->negative = true;
		}
		else if (floatString[charDigitIter] == '.')
		{
			multifloat->decimal = mpfloatDigitIter;
			FoundDecimal = true;
			NonZeroLeadingDigit = true;
		}
		else
		{
			uint8_t DigitToAdd = floatString[charDigitIter] - 48;
			if (!NonZeroLeadingDigit)
			{
				if (DigitToAdd)
				{
					NonZeroLeadingDigit = true;
				}
				else
				{
					continue;
				}
			}
			multifloat->digits[mpfloatDigitIter++] = DigitToAdd;
		}
	}

	if (multifloat->decimal == 0 && !FoundDecimal)
	{
		multifloat->decimal = charDigitIter - (int)(multifloat->negative);
	}
}

__device__ void mpfloat_clear(mpfloat *multifloat)
{
	if (multifloat->digits != nullptr)
	{
		for (int digitIter = 0; digitIter < mpfloat_precision; digitIter++)
		{
			multifloat->digits[digitIter] = 0;
		}
	}
	multifloat->decimal = 0;
	multifloat->negative = false;
}

__device__ void mpfloat_init(mpfloat *multifloat)
{
	multifloat->digits = new uint8_t[mpfloat_precision]();
	//hipMalloc(&multifloat->digits, mpfloat_precision * sizeof(uint8_t));
	mpfloat_clear(multifloat);
}

__device__ void mpfloat_shift_right_insert(mpfloat *multifloat, uint8_t digit)
{
	uint8_t ReplaceDigit = digit;
	uint8_t HolderDigit;

	for (uint32_t digitIter = 0; digitIter < mpfloat_precision; digitIter++)
	{
		HolderDigit = multifloat->digits[digitIter];
		multifloat->digits[digitIter] = ReplaceDigit;
		ReplaceDigit = HolderDigit;
	}

	if (digit)
	{
		multifloat->decimal++;
	}
}

void mpfloat_shift_right_insert_host(mpfloat *multifloat, uint8_t digit, bool IncrementDecimal)
{
	uint8_t ReplaceDigit = digit;
	uint8_t HolderDigit;

	for (uint32_t digitIter = 0; digitIter < mpfloat_precision; digitIter++)
	{
		HolderDigit = multifloat->digits[digitIter];
		multifloat->digits[digitIter] = ReplaceDigit;
		ReplaceDigit = HolderDigit;
	}

	if (IncrementDecimal)
	{
		multifloat->decimal++;
	}
}

__device__  void mpfloat_pack_left(mpfloat *multifloat)
{
	uint32_t NumLeadingZeroes = 0;
	for (int digitIter = 0; digitIter < mpfloat_precision; digitIter++)
	{
		if (multifloat->digits[digitIter])
		{
			break;
		}
		NumLeadingZeroes++;
	}
	multifloat->decimal -= NumLeadingZeroes;

	for (int digitIter = 0; digitIter < (mpfloat_precision - NumLeadingZeroes); digitIter++)
	{
		multifloat->digits[digitIter] = multifloat->digits[digitIter + NumLeadingZeroes];
	}
}

void mpfloat_pack_left_host(mpfloat *multifloat)
{
	uint32_t NumLeadingZeroes = 0;
	for (int digitIter = 0; digitIter < mpfloat_precision; digitIter++)
	{
		if (multifloat->digits[digitIter])
		{
			break;
		}
		NumLeadingZeroes++;
	}
	multifloat->decimal -= NumLeadingZeroes;

	for (int digitIter = 0; digitIter < (mpfloat_precision - NumLeadingZeroes); digitIter++)
	{
		multifloat->digits[digitIter] = multifloat->digits[digitIter + NumLeadingZeroes];
	}
}

__device__ int8_t mpfloat_cmp(mpfloat a, mpfloat b)
{
	// a > b  -->  1
	// a == b -->  0
	// a < b  --> -1

	if (a.negative && !b.negative)
	{
		return -1;
	}
	else if (b.negative && !a.negative)
	{
		return 1;
	}

	int8_t result = 0;

	if (a.decimal > b.decimal)
	{
		result = 1;
	}
	else if (a.decimal < b.decimal)
	{
		result = -1;
	}
	else // a.decimal == b.decimal
	{
		for (uint32_t digitIter = 0; digitIter < mpfloat_precision; digitIter++)
		{
			if (a.digits[digitIter] > b.digits[digitIter])
			{
				result = 1;
				break;
			}
			else if (a.digits[digitIter] < b.digits[digitIter])
			{
				result = -1;
				break;
			}
		}
	}

	if (b.negative && a.negative)
	{
		result *= -1;
	}

	return result;
}

int8_t mpfloat_cmp_host(mpfloat a, mpfloat b)
{
	// a > b  -->  1
	// a == b -->  0
	// a < b  --> -1

	if (a.negative && !b.negative)
	{
		return -1;
	}
	else if (b.negative && !a.negative)
	{
		return 1;
	}

	int8_t result = 0;

	if (a.decimal > b.decimal)
	{
		result = 1;
	}
	else if (a.decimal < b.decimal)
	{
		result = -1;
	}
	else // a.decimal == b.decimal
	{
		for (uint32_t digitIter = 0; digitIter < mpfloat_precision; digitIter++)
		{
			if (a.digits[digitIter] > b.digits[digitIter])
			{
				result = 1;
				break;
			}
			else if (a.digits[digitIter] < b.digits[digitIter])
			{
				result = -1;
				break;
			}
		}
	}

	if (b.negative && a.negative)
	{
		result *= -1;
	}

	return result;
}

__device__ int8_t mpfloat_cmp_ui(mpfloat a, uint32_t b)
{
	int8_t result;
	mpfloat tempfloat;
	mpfloat_init(&tempfloat);
	mpfloat_set_uint32(b, &tempfloat);
	result = mpfloat_cmp(a, tempfloat);
	mpfloat_free(tempfloat);
	return result;
}

uint32_t mpfloat_get_num_decimals_host(mpfloat multifloat)
{
	return mpfloat_get_num_digits_host(multifloat) - multifloat.decimal;
}

__device__  uint32_t mpfloat_get_num_decimals(mpfloat multifloat)
{
	return mpfloat_get_num_digits(multifloat) - multifloat.decimal;
}

uint32_t mpfloat_get_num_digits_host(mpfloat multifloat)
{
	uint32_t digitIter;

	//for (digitIter = (mpfloat_precision - 1); !multifloat.digits[digitIter]; digitIter--){}
	digitIter = mpfloat_precision - 1;
	while (true)
	{
		if (multifloat.digits[digitIter])
		{
			break;
		}

		if (digitIter == (multifloat.decimal - 1))
		{
			break;
		}
		
		if (digitIter == 0)
		{
			break;
		}
		digitIter--;
	}
	//for (digitIter = (mpfloat_precision - 1); !multifloat.digits[digitIter]; digitIter--) {}

	return digitIter + 1;
}

__device__  uint32_t mpfloat_get_num_digits(mpfloat multifloat)
{
	int32_t digitIter;

	//for (digitIter = (mpfloat_precision - 1); !multifloat.digits[digitIter]; digitIter--){}
	digitIter = mpfloat_precision - 1;
	while (digitIter >= 0)
	{
		if (multifloat.digits[digitIter])
		{
			break;
		}
		digitIter--;
	}
	//for (digitIter = (mpfloat_precision - 1); !multifloat.digits[digitIter]; digitIter--) {}

	return digitIter + 1;
}

__device__ void mpfloat_mul_ui(mpfloat a, uint32_t b, mpfloat *result)
{
	mpfloat tempfloat;
	mpfloat_init(&tempfloat);
	mpfloat_set_uint32(b, &tempfloat);

	mpfloat_mul(a, tempfloat, result);

	mpfloat_free(tempfloat);
}

__device__ void mpfloat_mul(mpfloat a, mpfloat b, mpfloat *result)
{
	char buff[100];
	mpfloat tempfloat;
	mpfloat resultCopy;
	mpfloat_init(&tempfloat);
	mpfloat_init(&resultCopy);



	int32_t DecimalOffset = b.decimal - a.decimal;
	uint8_t Carry = 0;

	uint32_t aDecimalCount = mpfloat_get_num_decimals(a);
	uint32_t aDigitCount = mpfloat_get_num_digits(a);

	uint32_t bDecimalCount = mpfloat_get_num_decimals(b);
	uint32_t bDigitCount = mpfloat_get_num_digits(b);

	if (bDigitCount == 0 || aDigitCount == 0)
	{
		// multiply by zero is zero
		mpfloat_clear(result);
		mpfloat_free(tempfloat);
		mpfloat_free(resultCopy);
		return;
	}

	uint32_t productDecimalCount = aDecimalCount + bDecimalCount;
	uint32_t tempfloatDecimalsAdded;
	int32_t aDigitIter;
	int32_t bDigitIter;
	uint8_t product = 0;


	for (aDigitIter = aDigitCount - 1; aDigitIter >= 0; aDigitIter--)
	{
		if (a.digits[aDigitIter])
		{
			mpfloat_clear(&tempfloat);
			tempfloatDecimalsAdded = 0;
			for (bDigitIter = bDigitCount - 1; bDigitIter >= 0; bDigitIter--)
			{
				product = a.digits[aDigitIter] * b.digits[bDigitIter] + Carry;
				tempfloat.digits[bDigitIter] = product % 10;
				Carry = (product - tempfloat.digits[bDigitIter]) / 10;
				tempfloatDecimalsAdded++;
			}
			if (Carry)
			{
				tempfloatDecimalsAdded++;
				mpfloat_shift_right_insert(&tempfloat, Carry);
				Carry = 0;
			}
			for (; tempfloatDecimalsAdded < productDecimalCount; tempfloatDecimalsAdded++)
			{
				mpfloat_shift_right_insert(&tempfloat, 0);
			}
			tempfloat.decimal = tempfloatDecimalsAdded - productDecimalCount;
			mpfloat_set_mpfloat(&resultCopy, *result);
			mpfloat_add(tempfloat, resultCopy, result);

			mpfloat_to_str(tempfloat, buff, 99);
			printf("tempfloat: %s \n", buff);
			mpfloat_to_str(resultCopy, buff, 99);
			printf("resultCopy: %s \n", buff);
			mpfloat_to_str(*result, buff, 99);
			printf("result: %s \n------------\n", buff);
		}
		productDecimalCount--;
	}

	result->negative = (a.negative != b.negative);

	mpfloat_free(tempfloat);
	mpfloat_free(resultCopy);
}

void mpfloat_mul_ui_host(mpfloat a, uint32_t b, mpfloat *result)
{

}

void mpfloat_mul_host(mpfloat a, mpfloat b, mpfloat *result)
{
	mpfloat tempfloat;
	mpfloat resultCopy;
	mpfloat_init_host(&tempfloat);
	mpfloat_init_host(&resultCopy);

	int32_t DecimalOffset = b.decimal - a.decimal;
	uint8_t Carry = 0;

	uint32_t aDecimalCount = mpfloat_get_num_decimals_host(a);
	uint32_t aDigitCount = mpfloat_get_num_digits_host(a);

	uint32_t bDecimalCount = mpfloat_get_num_decimals_host(b);
	uint32_t bDigitCount = mpfloat_get_num_digits_host(b);

	if (bDigitCount == 0 || aDigitCount == 0)
	{
		// multiply by zero is zero
		mpfloat_clear_host(result);
		mpfloat_free_host(tempfloat);
		mpfloat_free_host(resultCopy);
		return;
	}

	int32_t productDecimalCount = aDecimalCount + bDecimalCount;
	int32_t tempfloatDecimalsAdded;
	int32_t aDigitIter;
	int32_t bDigitIter;
	uint8_t product = 0;


	for (aDigitIter = aDigitCount - 1; aDigitIter >= 0; aDigitIter--)
	{
		if (a.digits[aDigitIter])
		{
			mpfloat_clear_host(&tempfloat);
			tempfloatDecimalsAdded = 0;
			for (bDigitIter = bDigitCount - 1; bDigitIter >= 0; bDigitIter--)
			{
				product = a.digits[aDigitIter] * b.digits[bDigitIter] + Carry;
				tempfloat.digits[bDigitIter] = product % 10;
				Carry = (product - tempfloat.digits[bDigitIter]) / 10;
				tempfloatDecimalsAdded++;
			}
			if (Carry)
			{
				tempfloatDecimalsAdded++;
				mpfloat_shift_right_insert_host(&tempfloat, Carry, true);
				Carry = 0;
			}
			for (; tempfloatDecimalsAdded < productDecimalCount; tempfloatDecimalsAdded++)
			{
				mpfloat_shift_right_insert_host(&tempfloat, 0, true);
			}
			tempfloat.decimal = tempfloatDecimalsAdded - productDecimalCount;
			mpfloat_set_mpfloat_host(&resultCopy, *result);
			mpfloat_add_host(tempfloat, resultCopy, result);
		}
		productDecimalCount--;
	}

	result->negative = (a.negative != b.negative);

	mpfloat_free_host(tempfloat);
	mpfloat_free_host(resultCopy);
}


__device__  void mpfloat_sub(mpfloat a, mpfloat b, mpfloat *result)
{
	bool bIsGreater = (mpfloat_cmp(a, b) == -1);
	if (a.negative && !b.negative)
	{
		// -a - b == -(a + b)
		b.negative = true;
		mpfloat_add(a, b, result);
		return;
	}
	else if (b.negative && !a.negative)
	{
		// a - (-b) == a + b
		mpfloat_add(a, b, result);
		return;
	}
	else if ((a.negative && b.negative) || bIsGreater)
	{
		// -a - (-b) == -a + b == b - a
		mpfloat temp;
		temp.decimal = a.decimal;
		temp.digits = a.digits;
		temp.negative = a.negative;

		a.decimal = b.decimal;
		a.digits = b.digits;
		a.negative = b.negative;

		b.decimal = temp.decimal;
		b.digits = temp.digits;
		b.negative = temp.negative;
	}



	int32_t DecimalOffset = b.decimal - a.decimal;
	uint32_t DigitIndex;
	int32_t Difference;
	if (a.decimal > b.decimal)
	{
		result->decimal = a.decimal;
	}
	else
	{
		result->decimal = b.decimal;
	}

	for (uint32_t digitIter = 0; digitIter < mpfloat_precision; digitIter++)
	{
		DigitIndex = mpfloat_precision - (digitIter + 1);
		if ((DigitIndex + DecimalOffset) >= mpfloat_precision)
		{
			continue;
		}

		Difference = a.digits[DigitIndex] - b.digits[DigitIndex + DecimalOffset];
		if (Difference < 0)
		{
			if (result->negative)
			{
				Difference *= -1;
			}
			else
			{
				for (int32_t BorrowIndex = (DigitIndex - 1); BorrowIndex >= 0; BorrowIndex--)
				{
					if (a.digits[BorrowIndex])
					{
						a.digits[BorrowIndex]--;
						for (uint32_t BorrowedDigitIter = 1; BorrowedDigitIter < (DigitIndex - BorrowIndex); BorrowedDigitIter++)
						{
							a.digits[BorrowIndex + BorrowedDigitIter] = 9;
						}
						Difference += 10;
						break;
					}
				}
				if (Difference < 0)
				{
					result->negative = true;
					Difference *= -1;
				}
			}
		}

		result->digits[DigitIndex] = Difference;

	}

	if (DecimalOffset < 0)
	{
		uint32_t AbsOffset = DecimalOffset * -1;
		for (uint32_t LeftOverDigitIter = 0; LeftOverDigitIter < AbsOffset; LeftOverDigitIter++)
		{
			result->digits[LeftOverDigitIter] = a.digits[LeftOverDigitIter];
		}
	}

	if (bIsGreater)
	{
		result->negative = true;
	}

	if (!result->digits[0])
	{
		mpfloat_pack_left(result);
	}

}

void mpfloat_sub_host(mpfloat a, mpfloat b, mpfloat *result)
{
	bool bIsGreater = (mpfloat_cmp_host(a, b) == -1);
	if (a.negative && !b.negative)
	{
		// -a - b == -(a + b)
		b.negative = true;
		mpfloat_add_host(a, b, result);
		return;
	}
	else if (b.negative && !a.negative)
	{
		// a - (-b) == a + b
		mpfloat_add_host(a, b, result);
		return;
	}
	else if ((a.negative && b.negative) || bIsGreater)
	{
		// -a - (-b) == -a + b == b - a
		mpfloat temp;
		temp.decimal = a.decimal;
		temp.digits = a.digits;
		temp.negative = a.negative;

		a.decimal = b.decimal;
		a.digits = b.digits;
		a.negative = b.negative;

		b.decimal = temp.decimal;
		b.digits = temp.digits;
		b.negative = temp.negative;
	}



	int32_t DecimalOffset = b.decimal - a.decimal;
	uint32_t DigitIndex;
	int32_t Difference;
	if (a.decimal > b.decimal)
	{
		result->decimal = a.decimal;
	}
	else
	{
		result->decimal = b.decimal;
	}

	for (uint32_t digitIter = 0; digitIter < mpfloat_precision; digitIter++)
	{
		DigitIndex = mpfloat_precision - (digitIter + 1);
		if ((DigitIndex + DecimalOffset) >= mpfloat_precision)
		{
			continue;
		}

		Difference = a.digits[DigitIndex] - b.digits[DigitIndex + DecimalOffset];
		if (Difference < 0)
		{
			if (result->negative)
			{
				Difference *= -1;
			}
			else
			{
				for (int32_t BorrowIndex = (DigitIndex - 1); BorrowIndex >= 0; BorrowIndex--)
				{
					if (a.digits[BorrowIndex])
					{
						a.digits[BorrowIndex]--;
						for (uint32_t BorrowedDigitIter = 1; BorrowedDigitIter < (DigitIndex - BorrowIndex); BorrowedDigitIter++)
						{
							a.digits[BorrowIndex + BorrowedDigitIter] = 9;
						}
						Difference += 10;
						break;
					}
				}
				if (Difference < 0)
				{
					result->negative = true;
					Difference *= -1;
				}
			}
		}

		result->digits[DigitIndex] = Difference;

	}

	if (DecimalOffset < 0)
	{
		uint32_t AbsOffset = DecimalOffset * -1;
		for (uint32_t LeftOverDigitIter = 0; LeftOverDigitIter < AbsOffset; LeftOverDigitIter++)
		{
			result->digits[LeftOverDigitIter] = a.digits[LeftOverDigitIter];
		}
	}

	if (bIsGreater)
	{
		result->negative = true;
	}

	if (!result->digits[0])
	{
		mpfloat_pack_left_host(result);
	}

}

__device__  void mpfloat_add(mpfloat a, mpfloat b, mpfloat *result)
{
	if (a.negative && !b.negative)
	{
		// -a + b == b - a
		a.negative = false;
		mpfloat_sub(b, a, result);
		return;
	}
	else if (b.negative && !a.negative)
	{
		// a + (-b) == a - b
		b.negative = false;
		mpfloat_sub(a, b, result);
		return;
	}
	else if (a.negative && b.negative)
	{
		// -a + (-b) == -(a + b)

		result->negative = true;
	}
	char *msg = new char[500];
	int32_t DecimalOffset = b.decimal - a.decimal;
	uint8_t Sum;
	uint8_t Carry = 0;
	uint32_t DigitIndex;

	result->decimal = a.decimal;

	for (uint32_t digitIter = 0; digitIter < mpfloat_precision; digitIter++)
	{
		DigitIndex = mpfloat_precision - (digitIter + 1);
		if ((DigitIndex + DecimalOffset) >= mpfloat_precision)
		{
			continue;
		}
		else if ((DigitIndex + DecimalOffset) < 0)
		{
			break;
		}
		Sum = a.digits[DigitIndex] + b.digits[DigitIndex + DecimalOffset] + (int)Carry;
		if (Sum >= 10)
		{
			Carry = 1;
			Sum -= 10;
		}
		else
		{
			Carry = 0;
		}
		result->digits[DigitIndex] = Sum;
	}

	if (DecimalOffset > 0)
	{
		for (uint32_t LeftOverDigitIter = 0; LeftOverDigitIter < DecimalOffset; LeftOverDigitIter++)
		{
			mpfloat_shift_right_insert(result, b.digits[DecimalOffset - (LeftOverDigitIter + 1)]);
		}
	}
	else if (DecimalOffset < 0)
	{
		uint32_t AbsOffset = DecimalOffset * -1;
		for (uint32_t LeftOverDigitIter = 0; LeftOverDigitIter < AbsOffset; LeftOverDigitIter++)
		{
			result->digits[LeftOverDigitIter] = a.digits[LeftOverDigitIter];
		}
	}
}

void mpfloat_add_host(mpfloat a, mpfloat b, mpfloat *result)
{
	if (a.negative && !b.negative)
	{
		// -a + b == b - a
		a.negative = false;
		//mpfloat_sub(b, a, result);
		return;
	}
	else if (b.negative && !a.negative)
	{
		// a + (-b) == a - b
		b.negative = false;
		//mpfloat_sub(a, b, result);
		return;
	}
	else if (a.negative && b.negative)
	{
		// -a + (-b) == -(a + b)

		result->negative = true;
	}

	int32_t DecimalOffset = b.decimal - a.decimal;
	uint8_t Sum;
	uint8_t Carry = 0;
	int32_t DigitIndex;
	uint32_t CarryDigitsAdded = 0;
	int32_t digitIter = 0;

	result->decimal = a.decimal;

	for (digitIter = mpfloat_precision-1; digitIter >= 0; digitIter--)
	{
		if ((digitIter + DecimalOffset) < 0)
		{
			break;
		}
		else if ((digitIter + DecimalOffset) >= (mpfloat_precision-1))
		{
			continue;
		}

		Sum = a.digits[digitIter] + b.digits[digitIter + DecimalOffset] + (int)Carry;
		if (Sum >= 10)
		{
			Carry = 1;
			Sum -= 10;
		}
		else
		{
			Carry = 0;
		}
		result->digits[digitIter] = Sum;
	}

	while (Carry)
	{
		if ((digitIter + DecimalOffset) >= 0)
		{
			if (DecimalOffset > 0)
			{
				Sum = b.digits[digitIter + DecimalOffset] + Carry;
			}
			else if (DecimalOffset < 0)
			{
				Sum = a.digits[digitIter + DecimalOffset] + Carry;
			}
			else
			{
				Sum = Carry;
			}
			
		}
		else if (DecimalOffset < 0 && digitIter >= 0)
		{
			Sum = a.digits[digitIter] + Carry;
		}
		else
		{
			Sum = Carry;
		}
		
		if (Sum >= 10)
		{
			Carry = 1;
			Sum -= 10;
		}
		else
		{
			Carry = 0;
		}

		if (digitIter >= 0)
		{
			result->digits[digitIter] = Sum;	
		}
		else
		{
			mpfloat_shift_right_insert_host(result, Sum, true);
		}

		digitIter--;
		CarryDigitsAdded++;
	}

	if (DecimalOffset > 0)
	{
		for (uint32_t LeftOverDigitIter = CarryDigitsAdded; LeftOverDigitIter < DecimalOffset; LeftOverDigitIter++)
		{
			mpfloat_shift_right_insert_host(result, b.digits[DecimalOffset - (LeftOverDigitIter + 1)], true);
		}
	}
	else if (DecimalOffset < 0)
	{
		uint32_t AbsOffset = DecimalOffset * -1;
		for (uint32_t LeftOverDigitIter = CarryDigitsAdded; LeftOverDigitIter < AbsOffset; LeftOverDigitIter++)
		{
			result->digits[LeftOverDigitIter] = a.digits[LeftOverDigitIter];
		}
	}
}

__device__ void mpfloat_set_mpfloat(mpfloat *a, mpfloat b)
{
	a->decimal = b.decimal;
	a->negative = b.negative;
	memcpy(a->digits, b.digits, mpfloat_precision * sizeof(uint8_t));
	//hipMemcpy(a->digits, b.digits, mpfloat_precision * sizeof(uint8_t), hipMemcpyHostToHost);
}

void mpfloat_set_mpfloat_host(mpfloat *a, mpfloat b)
{
	a->decimal = b.decimal;
	a->negative = b.negative;
	memcpy(a->digits, b.digits, mpfloat_precision * sizeof(uint8_t));
	//hipMemcpy(a->digits, b.digits, mpfloat_precision * sizeof(uint8_t), hipMemcpyHostToHost);
}

__device__ void mpfloat_to_double(mpfloat multifloat, double *result)
{
	*result = 0;
	uint64_t NumDigits;
	mpfloat_get_extent(multifloat, &NumDigits);
	double digitMagnitude = powf(10.0, multifloat.decimal - 1);
	int test = 0;
	for (uint32_t DigitIter = 0; DigitIter < NumDigits; DigitIter++)
	{
		*result += digitMagnitude * (double)(multifloat.digits[DigitIter]);
		digitMagnitude /= 10;
	}
	if (multifloat.negative)
	{
		*result *= -1;
	}
}

void mpfloat_set_mpir(mpfloat *multifloat, mpf_t mpirfloat)
{
	mp_exp_t exponent;
	char temp_buffer[200];
	mpf_get_str(temp_buffer, &exponent, 10, 90, mpirfloat);

	for (int charDigitIter = 0; charDigitIter < strlen(temp_buffer); charDigitIter++)
	{
		if (charDigitIter == 0 && temp_buffer[charDigitIter] == '-')
		{
			multifloat->negative = true;
		}
		else
		{
			multifloat->digits[charDigitIter - multifloat->negative] = temp_buffer[charDigitIter] - 48;
		}
	}

	multifloat->decimal = exponent;
}

void mpfloat_set_double_host(double a, mpfloat *result)
{
	char doubleString[128];
	mpfloat_clear_host(result);
	sprintf(doubleString, "%f", a);
	size_t stringLength = strlen(doubleString);
	uint32_t charIter;
	uint32_t digitIter = 0;
	bool gotDecimalPoint = false;
	bool gotNegative = false;

	for (charIter = 0; charIter < stringLength; charIter++)
	{
		if (doubleString[charIter] == '.')
		{
			if (a >= 1 || (a <= -1))
			{
				result->decimal = (charIter - gotNegative);
			}
			gotDecimalPoint = true;
		}
		else if (doubleString[charIter] == '-')
		{
			result->negative = true;
			gotNegative = true;
		}
		else if (digitIter == 0 && doubleString[charIter] == '0')
		{
			continue;
		}
		else
		{
			result->digits[digitIter++] = ((uint8_t)doubleString[charIter]) - 48;
		}

	}
}

__device__ void mpfloat_set_uint32(uint32_t a, mpfloat *result)
{
	mpfloat_clear(result);
	if (a == 0)
	{
		return;
	}

	uint32_t Reference = a;
	int InsertIndex = 0;
	uint32_t DivVal = 1000000000;
	bool FoundDigit = false;
	uint32_t digitValue;

	while (DivVal != 0)
	{
		digitValue = Reference / DivVal;
		if (digitValue != 0)
		{
			FoundDigit = true;
		}

		if (FoundDigit)
		{
			result->digits[InsertIndex++] = digitValue;
		}

		Reference %= DivVal;
		DivVal /= 10;
	}

	result->decimal = InsertIndex;
}

__device__ void mpfloat_get_extent(mpfloat a, uint64_t *extent)
{
	int decimalIter = 0;
	for (decimalIter = 0; decimalIter < mpfloat_precision; decimalIter++)
	{
		if (a.digits[(mpfloat_precision - 1) - decimalIter] != 0)
		{
			break;
		}

		if ((mpfloat_precision - 1) - decimalIter < a.decimal)
		{
			*extent = 0;
			return;
		}
	}

	*extent = (mpfloat_precision - decimalIter);
}

void mpfloat_get_extent_host(mpfloat a, uint64_t *extent)
{
	uint32_t decimalIter = 0;
	for (decimalIter = 0; decimalIter < mpfloat_precision; decimalIter++)
	{
		if (a.digits[(mpfloat_precision - 1) - decimalIter] != 0)
		{
			break;
		}

		if ((mpfloat_precision - 1) - decimalIter < a.decimal)
		{
			*extent = 0;
			return;
		}
	}

	*extent = (mpfloat_precision - decimalIter);
}

__device__ void mpfloat_to_str(mpfloat multifloat, char *buffer, uint32_t length)
{
	uint32_t CharIndexIter = 0;
	uint64_t decimalPlaces;
	mpfloat_get_extent(multifloat, &decimalPlaces);
	uint64_t printedDecimals = 0;
	bool GotDecimalPoint = false;

	if (multifloat.negative)
	{
		buffer[CharIndexIter++] = '-';
	}

	if (multifloat.decimal == 0)
	{
		buffer[CharIndexIter++] = '0';
	}

	for (int digitIter = 0; digitIter < mpfloat_precision; digitIter++)
	{
		if (digitIter == multifloat.decimal)
		{
			buffer[CharIndexIter++] = '.';
			GotDecimalPoint = true;
			if (CharIndexIter >= (length - 2))
			{
				break;
			}
		}

		buffer[CharIndexIter++] = multifloat.digits[digitIter] + 48;
		
		if (GotDecimalPoint)
		{
			printedDecimals++;
			if (printedDecimals >= decimalPlaces)
			{
				break;
			}
		}

		if (CharIndexIter >= (length - 2))
		{
			break;
		}
	}

	buffer[CharIndexIter] = '\0';
}

void mpfloat_to_str_host(mpfloat multifloat, char *buffer, uint32_t length)
{
	uint32_t CharIndexIter = 0;
	uint32_t decimalPlaces = mpfloat_get_num_decimals_host(multifloat);
	uint64_t printedDecimals = 0;
	bool GotDecimalPoint = false;

	if (multifloat.negative)
	{
		buffer[CharIndexIter++] = '-';
	}

	if (multifloat.decimal == 0)
	{
		buffer[CharIndexIter++] = '0';

		if (decimalPlaces == 0)
		{
			buffer[CharIndexIter] = '\0';
			return;
		}
	}

	for (int digitIter = 0; digitIter < mpfloat_precision; digitIter++)
	{
		if (digitIter == multifloat.decimal)
		{
			buffer[CharIndexIter++] = '.';
			GotDecimalPoint = true;
			if (CharIndexIter >= (length - 2))
			{
				break;
			}
		}

		buffer[CharIndexIter++] = multifloat.digits[digitIter] + 48;

		if (GotDecimalPoint)
		{
			printedDecimals++;
			if (printedDecimals >= decimalPlaces)
			{
				break;
			}
		}

		if (CharIndexIter >= (length - 2))
		{
			break;
		}
	}

	buffer[CharIndexIter] = '\0';
}

void print_mpfloat(mpfloat multifloat)
{
	uint64_t decimalPlaces;
	mpfloat_get_extent_host(multifloat, &decimalPlaces);
	uint64_t printedDecimals = 0;

	if (multifloat.negative)
	{
		fprintf(stdout, "-");
	}

	for (int digitIter = 0; digitIter < mpfloat_precision; digitIter++)
	{
		if (digitIter == multifloat.decimal)
		{
			fprintf(stdout, ".%d", multifloat.digits[digitIter]);
			printedDecimals++;
		}
		else
		{
			fprintf(stdout, "%d", multifloat.digits[digitIter]);
		}

		if (printedDecimals)
		{
			printedDecimals++;
			if (printedDecimals >= decimalPlaces)
			{
				break;
			}
		}

	}
}



#define DEBUG_RETURN return;

#define MEAN 0
#define MODE 1
#define MEDIAN 2

__device__ 	void getBestIteration(uint32_t *bestIteration, double *bestMagnitude, uint32_t *IterationSamples, double *MagnitudeSamples, uint32_t NumSamples, uint32_t statistic)
{

	switch (statistic)
	{
	case MODE:

		break;
	}
}

__global__ void doFractal(mpfloat leftX, mpfloat topI, mpfloat pixelCoordinateDelta, mpfloat superSampleCoordinateDelta, int frameWidth, int MaxIterations, int SampleRate, uint32_t *iterations, double *magnitudes, uint32_t Size, int precision)
{
	char *msg = new char[500];
	if (mpfloat_precision == 0)
	{
		mpfloat_precision = precision;
	}

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= Size)
	{
		return;
	}

	{

		mpfloat result;
		mpfloat_init(&result);
		mpfloat_to_str(leftX, msg, 498);
		printf("A: %s\n", msg);
		printf("A.decimal: %d\n", leftX.decimal);
		mpfloat_to_str(topI, msg, 498);
		printf("B: %s\n", msg);
		printf("B[0]: %u\n", topI.digits[0]);
		printf("B.decimal: %d\n", topI.decimal);
		mpfloat_add(topI, leftX, &result);
		mpfloat_to_str(result, msg, 498);
		printf("result: %s\n", msg);
	}
	DEBUG_RETURN

	mpfloat CurrentX;
	mpfloat_init(&CurrentX);
	mpfloat CurrentI;
	mpfloat_init(&CurrentI);
	mpfloat SuperI;
	mpfloat_init(&SuperI);
	mpfloat SuperX;
	mpfloat_init(&SuperX);
	mpfloat cx;
	mpfloat_init(&cx);
	mpfloat ci;
	mpfloat_init(&ci);
	mpfloat zx;
	mpfloat_init(&zx);
	mpfloat zi;
	mpfloat_init(&zi);
	mpfloat Magnitude;
	mpfloat_init(&Magnitude);
	mpfloat zx_squared;
	mpfloat_init(&zx_squared);
	mpfloat zi_squared;
	mpfloat_init(&zi_squared);
	mpfloat tempzx;
	mpfloat_init(&tempzx);
	mpfloat tempzi;
	mpfloat_init(&tempzi);
	mpfloat RowPixelOffset;
	mpfloat_init(&RowPixelOffset);
	mpfloat tempfloat;
	mpfloat_init(&tempfloat);

	double MinIterationMagnitude = 0.0;
	int CellsCompleted = 0;
	int MinIteration = 0;
	int CurrentIterations = 0;
	bool PreviouslyPaused = false;
	bool DoFullCalculation = true;

	uint32_t *IterationSamples = new uint32_t[SampleRate*SampleRate];
	double *MagnitudeSamples = new double[SampleRate*SampleRate];

	uint32_t Row = idx / frameWidth;
	// Current I = topI - (rows * pixelcoordinatedelta)
	mpfloat_mul_ui(pixelCoordinateDelta, Row, &CurrentI);
	mpfloat_sub(topI, CurrentI, &CurrentI);

	

	uint32_t superSamplesCalculated = 0;
	double CurrentMagnitude;

	mpfloat_set_mpfloat(&SuperI, CurrentI);


	mpfloat_set_mpfloat(&CurrentX, leftX);

	for (int SuperRow = 0; SuperRow < SampleRate; SuperRow++)
	{
		mpfloat_set_mpfloat(&SuperX, CurrentX);

		for (int SuperCol = 0; SuperCol < SampleRate; SuperCol++)
		{
			mpfloat_set_mpfloat(&cx, SuperX);
			mpfloat_set_mpfloat(&ci, SuperI);
			mpfloat_set_mpfloat(&zx, SuperX);
			mpfloat_set_mpfloat(&zi, SuperI);
			mpfloat_clear(&Magnitude);

			for (CurrentIterations = 0; CurrentIterations < MaxIterations; CurrentIterations++)
			{
				mpfloat_mul(zx, zx, &zx_squared);
				mpfloat_to_str(zx, msg, 499);
				printf("zx is: %s\n", msg);
				mpfloat_to_str(zx_squared, msg, 499);
				printf("zx_squared is: %s\n", msg);
				DEBUG_RETURN
				mpfloat_mul(zi, zi, &zi_squared);
				mpfloat_sub(zx_squared, zi_squared, &tempzx);
				mpfloat_sub(zx, zi, &tempzi);
				mpfloat_mul_ui(tempzi, 2, &tempfloat);

				mpfloat_add(tempzx, cx, &zx);
				mpfloat_add(tempfloat, ci, &zi);

				mpfloat_mul(zx, zx, &zx_squared);
				mpfloat_mul(zi, zi, &zi_squared);

				mpfloat_add(zx_squared, zi_squared, &Magnitude);

				if (Magnitude.decimal > 0)
				{
					if (Magnitude.decimal > 1 || Magnitude.digits[0] >= 4)
					{
						break;
					}
				}
			}
			mpfloat_to_double(Magnitude, &CurrentMagnitude);

			IterationSamples[superSamplesCalculated] = CurrentIterations;
			MagnitudeSamples[superSamplesCalculated] = CurrentMagnitude;

			mpfloat_add(SuperX, superSampleCoordinateDelta, &tempfloat);
			mpfloat_set_mpfloat(&SuperX, tempfloat);
		}
		mpfloat_sub(SuperI, superSampleCoordinateDelta, &tempfloat);
		mpfloat_set_mpfloat(&SuperI, tempfloat);

	}


	iterations[idx] = CurrentIterations;
	magnitudes[idx] = CurrentMagnitude;

	//getBestIteration(&iterations[idx], &magnitudes[idx], IterationSamples, MagnitudeSamples, (SampleRate*SampleRate), MODE);
	delete IterationSamples;
	delete MagnitudeSamples;
	mpfloat_free(CurrentX);
	mpfloat_free(SuperI);
	mpfloat_free(SuperX);
	mpfloat_free(CurrentI);
	mpfloat_free(cx);
	mpfloat_free(ci);
	mpfloat_free(zx);
	mpfloat_free(zi);
	mpfloat_free(Magnitude);
	mpfloat_free(zx_squared);
	mpfloat_free(zi_squared);
	mpfloat_free(tempzx);
	mpfloat_free(RowPixelOffset);
	mpfloat_free(tempfloat);
}
#if 0
void doFractal_host(mpfloat leftX, mpfloat topI, mpfloat pixelCoordinateDelta, mpfloat superSampleCoordinateDelta, int frameWidth, int MaxIterations, int SampleRate, uint32_t *iterations, double *magnitudes, uint32_t Size, int precision)
{
	if (mpfloat_precision == 0)
	{
		mpfloat_precision = precision;
	}

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= Size)
	{
		return;
	}

	mpfloat CurrentX;
	mpfloat_init(&CurrentX);
	mpfloat CurrentI;
	mpfloat_init(&CurrentI);
	mpfloat SuperI;
	mpfloat_init(&SuperI);
	mpfloat SuperX;
	mpfloat_init(&SuperX);
	mpfloat cx;
	mpfloat_init(&cx);
	mpfloat ci;
	mpfloat_init(&ci);
	mpfloat zx;
	mpfloat_init(&zx);
	mpfloat zi;
	mpfloat_init(&zi);
	mpfloat Magnitude;
	mpfloat_init(&Magnitude);
	mpfloat zx_squared;
	mpfloat_init(&zx_squared);
	mpfloat zi_squared;
	mpfloat_init(&zi_squared);
	mpfloat tempzx;
	mpfloat_init(&tempzx);
	mpfloat tempzi;
	mpfloat_init(&tempzi);
	mpfloat RowPixelOffset;
	mpfloat_init(&RowPixelOffset);
	mpfloat tempfloat;
	mpfloat_init(&tempfloat);

	double MinIterationMagnitude = 0.0;
	int CellsCompleted = 0;
	int MinIteration = 0;
	int CurrentIterations = 0;
	bool PreviouslyPaused = false;
	bool DoFullCalculation = true;

	uint32_t *IterationSamples = new uint32_t[SampleRate*SampleRate];
	double *MagnitudeSamples = new double[SampleRate*SampleRate];

	uint32_t Row = idx / frameWidth;
	// Current I = topI - (rows * pixelcoordinatedelta)
	mpfloat_mul_ui(pixelCoordinateDelta, Row, &CurrentI);
	mpfloat_sub(topI, CurrentI, &CurrentI);



	uint32_t superSamplesCalculated = 0;
	double CurrentMagnitude;

	mpfloat_set_mpfloat(&SuperI, CurrentI);


	mpfloat_set_mpfloat(&CurrentX, leftX);

	for (int SuperRow = 0; SuperRow < SampleRate; SuperRow++)
	{
		mpfloat_set_mpfloat(&SuperX, CurrentX);

		for (int SuperCol = 0; SuperCol < SampleRate; SuperCol++)
		{
			mpfloat_set_mpfloat(&cx, SuperX);
			mpfloat_set_mpfloat(&ci, SuperI);
			mpfloat_set_mpfloat(&zx, SuperX);
			mpfloat_set_mpfloat(&zi, SuperI);
			mpfloat_clear(&Magnitude);

			for (CurrentIterations = 0; CurrentIterations < MaxIterations; CurrentIterations++)
			{
				mpfloat_mul(zx, zx, &zx_squared);

				DEBUG_RETURN
				mpfloat_mul(zi, zi, &zi_squared);
				mpfloat_sub(zx_squared, zi_squared, &tempzx);
				mpfloat_sub(zx, zi, &tempzi);
				mpfloat_mul_ui(tempzi, 2, &tempfloat);

				mpfloat_add(tempzx, cx, &zx);
				mpfloat_add(tempfloat, ci, &zi);

				mpfloat_mul(zx, zx, &zx_squared);
				mpfloat_mul(zi, zi, &zi_squared);

				mpfloat_add(zx_squared, zi_squared, &Magnitude);

				if (Magnitude.decimal > 0)
				{
					if (Magnitude.decimal > 1 || Magnitude.digits[0] >= 4)
					{
						break;
					}
				}
			}
			mpfloat_to_double(Magnitude, &CurrentMagnitude);

			IterationSamples[superSamplesCalculated] = CurrentIterations;
			MagnitudeSamples[superSamplesCalculated] = CurrentMagnitude;

			mpfloat_add(SuperX, superSampleCoordinateDelta, &tempfloat);
			mpfloat_set_mpfloat(&SuperX, tempfloat);
		}
		mpfloat_sub(SuperI, superSampleCoordinateDelta, &tempfloat);
		mpfloat_set_mpfloat(&SuperI, tempfloat);

	}


	iterations[idx] = CurrentIterations;
	magnitudes[idx] = CurrentMagnitude;

	//getBestIteration(&iterations[idx], &magnitudes[idx], IterationSamples, MagnitudeSamples, (SampleRate*SampleRate), MODE);
	delete IterationSamples;
	delete MagnitudeSamples;
	mpfloat_free(CurrentX);
	mpfloat_free(SuperI);
	mpfloat_free(SuperX);
	mpfloat_free(CurrentI);
	mpfloat_free(cx);
	mpfloat_free(ci);
	mpfloat_free(zx);
	mpfloat_free(zi);
	mpfloat_free(Magnitude);
	mpfloat_free(zx_squared);
	mpfloat_free(zi_squared);
	mpfloat_free(tempzx);
	mpfloat_free(RowPixelOffset);
	mpfloat_free(tempfloat);
}
#endif

__device__ unsigned int dev_gx;
__device__ unsigned int dev_bx;


__global__ void cudaTesting()
{
	if (threadIdx.x == 0)
	{
		dev_gx = gridDim.x;
		dev_bx = blockDim.x;
	}
}
void CUDAFractalWorker(mpf_t leftX, mpf_t topI, mpf_t pixelCoordinateDelta, mpf_t SuperSampleCoordinateDelta, int frameWidth, int frameHeight, int MaxIterations, int SampleRate, int Precision, uint32_t *HostIterations, double *HostMagnitudes)
{
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}

	mpfloat_set_precision(Precision);

#if 0
	char buff[100];
	mpfloat multifloattest;
	mpfloat_init_host(&multifloattest);
	cudaCheckError();
	mpfloat_set_double(1.325, &multifloattest);
	double test;
	mpfloat_to_double(multifloattest, &test);
	fprintf(stdout, "%f\n\n", test);
	//mpfloat_to_str(multifloattest, buff, 100);
	mpfloat_free_host(multifloattest);
	return;
#endif
#if 1
	int blockSize;   // The launch configurator returned block size 
	int minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
	int gridSize;    // The actual grid size needed, based on input size 
	int arrayCount = frameWidth*frameHeight;
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, doFractal, 0, 0);
	// Round up according to array size 
	gridSize = (arrayCount + blockSize - 1) / blockSize;
	cudaCheckError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stdout, "Cuda Error: %d --> %s\n", hipGetLastError(), hipGetErrorString(cudaStatus));
		return;
	}

	fprintf(stdout, "GridSize: %d BlockSize: %d\n", gridSize, blockSize);




	mpfloat multifloat_leftX;
	mpfloat multifloat_topI;
	mpfloat multifloat_pixelDelta;
	mpfloat multifloat_superPixelDelta;
	mpfloat_init_host(&multifloat_leftX);
	mpfloat_set_mpir(&multifloat_leftX, leftX);
	mpfloat_init_host(&multifloat_topI);
	mpfloat_set_mpir(&multifloat_topI, topI);
	mpfloat_init_host(&multifloat_pixelDelta);
	mpfloat_set_mpir(&multifloat_pixelDelta, pixelCoordinateDelta);
	mpfloat_init_host(&multifloat_superPixelDelta);
	mpfloat_set_mpir(&multifloat_superPixelDelta, SuperSampleCoordinateDelta);
	cudaCheckError();

	uint32_t *DeviceIterations;
	double *DeviceMagnitudes;
	cudaStatus = hipMallocManaged(&DeviceIterations, arrayCount * sizeof(uint32_t));
	cudaCheckError();
	if (cudaStatus == hipSuccess)
	{
		cudaStatus = hipMallocManaged(&DeviceMagnitudes, arrayCount * sizeof(double));
		cudaCheckError();
	}

	//__global__ void doFractal(mpfloat leftX, mpfloat topI, mpfloat pixelCoordinateDelta, mpfloat superSampleCoordinateDelta, uint32_t *iterations, uint32_t Size)

	if (cudaStatus == hipSuccess)
	{
		//mpfloat_test_mul_host();
		unsigned int *gx;
		unsigned int *bx;
		cudaStatus = hipMallocManaged(gx, sizeof(unsigned int));
		cudaTesting << <gridSize, blockSize >> > ();

		

		//doFractal << <gridSize, blockSize >> > (multifloat_leftX, multifloat_topI, multifloat_pixelDelta, multifloat_superPixelDelta, frameWidth, MaxIterations, SampleRate, DeviceIterations, DeviceMagnitudes, arrayCount, Precision);

		//doFractal << <1, 1 >> > (multifloat_leftX, multifloat_topI, multifloat_pixelDelta, multifloat_superPixelDelta, frameWidth, MaxIterations, SampleRate, DeviceIterations, DeviceMagnitudes, arrayCount, Precision);

	}
	if (cudaStatus == hipSuccess)
	{
		cudaStatus = hipDeviceSynchronize();
		cudaCheckError();
	}

	if (cudaStatus == hipSuccess)
	{
		unsigned int hst_gx;
		hipMemcpyFromSymbol(&hst_gx, HIP_SYMBOL(&dev_gx), sizeof(hst_gx), 0, hipMemcpyDeviceToHost);
		fprintf(stdout, "Gridx: %d\n", hst_gx);

		cudaStatus = hipMemcpy(HostIterations, DeviceIterations, arrayCount * sizeof(uint32_t), hipMemcpyDeviceToHost);
		cudaCheckError();
		fprintf(stdout, "iter: %d\n", HostIterations[0]);
	}
	if (cudaStatus == hipSuccess)
	{
		cudaStatus = hipMemcpy(HostMagnitudes, DeviceMagnitudes, arrayCount * sizeof(double), hipMemcpyDeviceToHost);
		cudaCheckError();
	}

	hipFree(DeviceIterations);
	hipFree(DeviceMagnitudes);
	mpfloat_free_host(multifloat_leftX);
	mpfloat_free_host(multifloat_topI);
	mpfloat_free_host(multifloat_superPixelDelta);
	mpfloat_free_host(multifloat_pixelDelta);
#endif
	return;
}
